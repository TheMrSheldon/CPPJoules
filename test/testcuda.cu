#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "../include/cppJoules.h"

// CUDA kernel for matrix multiplication
__global__ void matMulKernel(float *A, float *B, float *C, int N)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;

    if (row < N && col < N)
    {
        for (int k = 0; k < N; ++k)
        {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

// Function to initialize matrices with random values
void initializeMatrix(float *mat, int N)
{
    for (int i = 0; i < N * N; ++i)
    {
        mat[i] = static_cast<float>(rand()) / RAND_MAX;
    }
}

int main()
{
    int N = 1024; // Size of the matrices (N x N)
    size_t size = N * N * sizeof(float);

    // Allocate host matrices
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    // Initialize host matrices
    initializeMatrix(h_A, N);
    initializeMatrix(h_B, N);

    // Allocate device matrices
    float *d_A;
    float *d_B;
    float *d_C;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // Copy host matrices to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Define block and grid dimensions
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x, (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    EnergyTracker tracker;
    // Launch the matrix multiplication kernel
    tracker.start();
    for (int i = 0; i < 10e3; i++)
    {
        matMulKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    }

    // Copy result matrix back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Free device matrices
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host matrices
    free(h_A);
    free(h_B);
    free(h_C);
    tracker.stop();
    tracker.calculate_energy();
    tracker.print_energy();

    printf("Matrix multiplication completed successfully.\n");

    return 0;
}
